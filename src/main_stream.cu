#include "stereo.cuh"


int main_stream()
{
	float elapseTime;
	const int streamSize = 5;
	hipEvent_t  start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	Mat imL = imread("imr.png", IMREAD_GRAYSCALE);//��ͼ
	Mat imR = imread("iml.png", IMREAD_GRAYSCALE);//��ͼ
	Mat imD = Mat::zeros(imL.size(), imL.type());//�Ӳ�ͼ
	printf("%d %d\n", imL.cols, imL.rows);
	if (imL.empty() || imR.empty()) {
		perror("load image\n");
		return -1;
	}
	uchar *data[3], *devData[streamSize][3];
	int dataSize = imL.cols*imL.rows, Row = imL.rows, Col = imL.cols;
	///////////��ʼ��ʱ
	HANDLE_ERROR(hipEventRecord(start, 0));
	//�ڴ����
	for (int i = 0; i < streamSize; i++) {
		for (int j = 0; j < 3; j++) {
			HANDLE_ERROR(hipMalloc((void**)&devData[i][j], dataSize/streamSize * sizeof(uchar)+WinSize*Col));
		}
	}
	//HANDLE_ERROR(hipEventRecord(start, 0));
	for (int i = 0; i < 3; i++) {
		HANDLE_ERROR(hipHostAlloc((void**)&data[i], dataSize*sizeof(uchar), hipHostMallocDefault));//page-locked host memory
	}
	fillData(imL, data[0]);
	fillData(imR, data[1]);

	//kernel stream
	hipStream_t  stream[streamSize];
	for (int i = 0; i < streamSize; i++) {
		HANDLE_ERROR(hipStreamCreate(&stream[i]));
	}
	int step = Row / streamSize, r = WinSize/2;
	for (int i = 0; i < streamSize; i++) {
		int beg, end, start;
		if (i == 0) {
			beg = 0;	end = step;    start = 0;
		}else if (i == streamSize - 1) {
			beg = i*step - r;   end = Row - r;   start = r;
		}else {
			beg = i*step - r;   end = (i + 1)*step;   start = r;
		}
		HANDLE_ERROR(hipMemcpyAsync(devData[i][0], data[0]+beg*Col, (end-beg)*Col*sizeof(uchar), hipMemcpyHostToDevice, stream[i]));
		HANDLE_ERROR(hipMemcpyAsync(devData[i][1], data[1]+beg*Col, (end-beg)*Col*sizeof(uchar), hipMemcpyHostToDevice, stream[i]));
		getDepthMap_stream<<<dim3(4, end-beg), dim3(128, 1), 0, stream[i] >>>(devData[i][0], devData[i][1], devData[i][2], Col, Row);
		HANDLE_ERROR(hipMemcpyAsync(data[2]+i*step*Col, devData[i][2]+start*Col, (end-beg-start)*Col*sizeof(uchar), hipMemcpyDeviceToHost, stream[i]));
	}
	//
	for (int i = 0; i < streamSize; i++) {
		HANDLE_ERROR(hipStreamSynchronize(stream[i]));
	}
	fillImage(data[2], imD);
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapseTime, start, stop));
	imshow("stereo", imD);
	imwrite("imD.png", imD);
	printf("time = %lfms\n", elapseTime);
	waitKey(0);
	return 0;
}