#include "stereo.cuh"


int main(int argc, char **argv)
{
	if (argc != 3) {
		printf("Parameter: imgleft, imgright\n");
		return 0;
	}
	float elapseTime;
	hipEvent_t  start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	Mat imL = imread("imr.png", IMREAD_GRAYSCALE);//��ͼ
	Mat imR = imread("iml.png", IMREAD_GRAYSCALE);//��ͼ
	Mat imD = Mat::zeros(imL.size(), imL.type());//�Ӳ�ͼ
	printf("%d %d\n", imL.cols, imL.rows);
	if (imL.empty() || imR.empty()) {
		perror("load image\n");
		return -1;
	}
	uchar *data[3], *devData[3];
	int dataSize = imL.cols*imL.rows;
	HANDLE_ERROR(hipEventRecord(start, 0));
	//�ڴ����
	for (int i = 0; i < 3; i++) {
		data[i] = (uchar*)malloc(dataSize * sizeof(uchar));
		HANDLE_ERROR(hipMalloc((void**)&devData[i], dataSize * sizeof(uchar)));
	}
	//Host to Device
	fillData(imL, data[0]);
	fillData(imR, data[1]);
	for (int i = 0; i < 2; i++) {
		HANDLE_ERROR(hipMemcpy(devData[i], data[i], sizeof(uchar)*dataSize, hipMemcpyHostToDevice));
	}
	//kernel
	getDepthMap_shared <<<375, 450>>> (devData[0], devData[1], devData[2], imL.cols, imL.rows);
	
	//Device to Host
	HANDLE_ERROR(hipMemcpy(data[2], devData[2], sizeof(uchar)*dataSize, hipMemcpyDeviceToHost));
	fillImage(data[2], imD);
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapseTime, start, stop));
	imshow("stereo", imD);
	imwrite("imD.png", imD);
	printf("time = %lfms\n", elapseTime);
	waitKey(0);
	return 0;
}