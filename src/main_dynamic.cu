#include "stereo.cuh"


int main_dynamic()
{
	float elapseTime;
	hipEvent_t  start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	Mat imL = imread("imr.png", IMREAD_GRAYSCALE);//��ͼ
	Mat imR = imread("iml.png", IMREAD_GRAYSCALE);//��ͼ
	Mat imD = Mat::zeros(imL.size(), imL.type());//�Ӳ�ͼ
	printf("%d %d\n", imL.cols, imL.rows);
	if (imL.empty() || imR.empty()) {
		perror("load image\n");
		return -1;
	}
	uchar *data[3], *devData[3];
	int dataSize = imL.cols*imL.rows;
	HANDLE_ERROR(hipEventRecord(start, 0));
	//�ڴ����
	for (int i = 0; i < 3; i++) {
		data[i] = (uchar*)malloc(dataSize * sizeof(uchar));
		HANDLE_ERROR(hipMalloc((void**)&devData[i], dataSize * sizeof(uchar)));
	}
	//Host to Device
	fillData(imL, data[0]);
	fillData(imR, data[1]);
	for (int i = 0; i < 2; i++) {
		HANDLE_ERROR(hipMemcpy(devData[i], data[i], sizeof(uchar)*dataSize, hipMemcpyHostToDevice));
	}
	//kernel
	//getDepthMap_Dynamic << <dim3(64, 64), dim3(8, 8) >> >(devData[0], devData[1], devData[2], imL.cols, imL.rows);
	//Device to Host
	HANDLE_ERROR(hipMemcpy(data[2], devData[2], sizeof(uchar)*dataSize, hipMemcpyDeviceToHost));
	fillImage(data[2], imD);
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapseTime, start, stop));
	imshow("stereo", imD);
	imwrite("imD.png", imD);
	printf("time = %lfms\n", elapseTime);
	waitKey(0);
	return 0;
}